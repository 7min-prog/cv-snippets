#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <Eigen/Dense>
#include <iostream>
#include "kernel.hpp"

#define CHECK(call)                                                            \
    {                                                                          \
        const hipError_t error = call;                                        \
        if (error != hipSuccess) {                                            \
            std::cout << "Error: " << __FILE__ << ":" << __LINE__ << std::endl \
                      << hipGetErrorString(error) << std::endl;               \
            exit(1);                                                           \
        }                                                                      \
    }

__global__ void add_vector_fixed_kernel(const MyVector *in1, const MyVector *in2, MyVector *out) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= VECLEN) return;
    out[i] = in1[i] + in2[i];
}

void add_vector_fixed(const MyVector &in1, const MyVector &in2, MyVector &out) {
    MyVector *d_in1, *d_in2, *d_out;
    size_t memsize = sizeof(MyVector);
    CHECK(hipMalloc((void **)&d_in1, memsize));
    CHECK(hipMalloc((void **)&d_in2, memsize));
    CHECK(hipMalloc((void **)&d_out, memsize));

    CHECK(hipMemcpy(d_in1, in1.data(), memsize, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_in2, in2.data(), memsize, hipMemcpyHostToDevice));

    int threadsPerBlock = 256;
    int blocksPerGrid = (VECLEN + threadsPerBlock - 1) / threadsPerBlock;
    add_vector_fixed_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_in1, d_in2, d_out);
    CHECK(hipGetLastError());

    CHECK(hipMemcpy(out.data(), d_out, memsize, hipMemcpyDeviceToHost));

    CHECK(hipFree(d_in1));
    CHECK(hipFree(d_in2));
    CHECK(hipFree(d_out));
}

__global__ void add_vector_kernel(const double *in1, const double *in2, double *out, const int n) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= n) return;
    out[i] = in1[i] + in2[i];
}

void add_vector(const Eigen::VectorXd &in1, const Eigen::VectorXd &in2, Eigen::VectorXd &out) {
    if(in1.rows() != in2.rows() || in1.rows() != out.rows()) return;

    int len = in1.rows();
    double *d_in1, *d_in2, *d_out;
    size_t memsize = sizeof(double) * len;
    CHECK(hipMalloc((void **)&d_in1, memsize));
    CHECK(hipMalloc((void **)&d_in2, memsize));
    CHECK(hipMalloc((void **)&d_out, memsize));

    CHECK(hipMemcpy(d_in1, in1.data(), memsize, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_in2, in2.data(), memsize, hipMemcpyHostToDevice));

    int threadsPerBlock = 256;
    int blocksPerGrid = (len + threadsPerBlock - 1) / threadsPerBlock;
    add_vector_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_in1, d_in2, d_out, len);
    CHECK(hipGetLastError());

    CHECK(hipMemcpy(out.data(), d_out, memsize, hipMemcpyDeviceToHost));

    CHECK(hipFree(d_in1));
    CHECK(hipFree(d_in2));
    CHECK(hipFree(d_out));
}
