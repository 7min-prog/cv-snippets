#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <opencv2/opencv.hpp>

#include <cmath>
#include <iostream>

#define CHECK(call)                                                      \
  {                                                                      \
    const hipError_t error = call;                                      \
    if (error != hipSuccess) {                                          \
      std::cout << "Error: " << __FILE__ << ":" << __LINE__ << std::endl \
                << hipGetErrorString(error) << std::endl;               \
      exit(1);                                                           \
    }                                                                    \
  }

int main() {
  cv::Mat img = cv::imread("lenna.png", 0);

  int NX = img.cols;
  int NY = img.rows;
  size_t size = sizeof(hipfftDoubleComplex) * NX * NY;

  hipfftDoubleComplex *input = (hipfftDoubleComplex *)malloc(size);
  for (int y = 0; y < NY; y++) {
    for (int x = 0; x < NX; x++) {
      input[NX * y + x] = make_hipDoubleComplex(img.at<uint8_t>(y, x), 0);
    }
  }

  hipfftHandle plan;
  hipfftPlan2d(&plan, NX, NY, HIPFFT_Z2Z);
  hipfftDoubleComplex *idata, *odata;

  CHECK(hipMalloc((void **)&idata, size));
  CHECK(hipMalloc((void **)&odata, size));
  CHECK(hipMemcpy(idata, input, size, hipMemcpyHostToDevice));

  hipfftExecZ2Z(plan, idata, odata, HIPFFT_FORWARD);
  hipfftDoubleComplex *freq = (hipfftDoubleComplex *)malloc(size);
  CHECK(hipMemcpy(freq, odata, size, hipMemcpyDeviceToHost));

  for (int y = 0; y < NY; y++) {
    for (int x = 0; x < NX; x++) {
      freq[NX * y + x].x /= (NX * NY);
      freq[NX * y + x].y /= (NX * NY);
    }
  }
  std::cout << freq[0].x << ", " << freq[0].y << std::endl;

  CHECK(hipMemcpy(idata, freq, size, hipMemcpyHostToDevice));
  hipfftExecZ2Z(plan, idata, odata, HIPFFT_BACKWARD);

  hipfftDoubleComplex *result = (hipfftDoubleComplex *)malloc(size);
  CHECK(hipMemcpy(result, odata, size, hipMemcpyDeviceToHost));

  hipfftDestroy(plan);
  hipFree(idata);
  hipFree(odata);

  std::cout << result[0].x << ", " << result[0].y << std::endl;
  cv::Mat res_img(NY, NX, CV_8U, cv::Scalar(0));
  for (int y = 0; y < NY; y++) {
    for (int x = 0; x < NX; x++) {
      float real = result[NX * y + x].x;
      float image = result[NX * y + x].y;
      res_img.at<uint8_t>(y, x) = (int)(real);
    }
  }
  cv::imwrite("test.png", res_img);

  free(input);
  free(freq);
  free(result);
}
