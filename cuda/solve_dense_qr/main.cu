#include <assert.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hipsolver.h>
#include <stdlib.h>
#include <iostream>

#define CUDACHECK(call)              \
  {                                  \
    const hipError_t status = call; \
    assert(status == hipSuccess);   \
  }
#define CUSLVCHECK(call)                       \
  {                                            \
    const hipsolverStatus_t status = call;      \
    assert(status == HIPSOLVER_STATUS_SUCCESS); \
  }
#define CUBLASCHECK(call)                    \
  {                                          \
    const hipblasStatus_t status = call;      \
    assert(status == HIPBLAS_STATUS_SUCCESS); \
  }

void printMatrix(int m, int n, const double *A, int lda) {
  for (int row = 0; row < m; row++) {
    for (int col = 0; col < n; col++) { std::cout << A[row + col * lda] << ",\t"; }
    std::cout << std::endl;
  }
}

int main(int argc, char *argv[]) {
  hipsolverHandle_t cusolverH = NULL;
  hipblasHandle_t cublasH = NULL;
  const int m = 3;
  const int lda = m;
  const int ldb = m;
  const int nrhs = 1;  // number of right hand side vectors

  /* | 1 2 3 |
   * A = | 4 5 6 |
   * | 2 1 1 |
   *
   * x = (1 1 1)'
   * b = (6 15 4)'
   */
  double A[lda * m] = {1.0, 4.0, 2.0, 2.0, 5.0, 1.0, 3.0, 6.0, 1.0};
  // double X[ldb*nrhs] = { 1.0, 1.0, 1.0}; // exact solution
  double B[ldb * nrhs] = {6.0, 15.0, 4.0};
  double XC[ldb * nrhs];  // solution matrix from GPU
  double *d_A = NULL;     // linear memory of GPU
  double *d_tau = NULL;   // linear memory of GPU
  double *d_B = NULL;
  int *devInfo = NULL;  // info in gpu (device copy)
  double *d_work = NULL;
  int lwork = 0;
  int info_gpu = 0;
  const double one = 1;
  std::cout << "A = (matlab base-1)" << std::endl;
  printMatrix(m, m, A, lda);
  std::cout << "=====" << std::endl;
  std::cout << "B = (matlab base-1)" << std::endl;
  printMatrix(m, nrhs, B, ldb);
  std::cout << "=====" << std::endl;
  // step 1: create cusolver/cublas handle
  CUSLVCHECK(hipsolverDnCreate(&cusolverH));
  CUBLASCHECK(hipblasCreate(&cublasH));

  // step 2: copy A and B to device
  CUDACHECK(hipMalloc((void **)&d_A, sizeof(double) * lda * m));
  CUDACHECK(hipMalloc((void **)&d_tau, sizeof(double) * m));
  CUDACHECK(hipMalloc((void **)&d_B, sizeof(double) * ldb * nrhs));
  CUDACHECK(hipMalloc((void **)&devInfo, sizeof(int)));
  CUDACHECK(hipMemcpy(d_A, A, sizeof(double) * lda * m, hipMemcpyHostToDevice));
  CUDACHECK(hipMemcpy(d_B, B, sizeof(double) * ldb * nrhs, hipMemcpyHostToDevice));

  // step 3: query working space of geqrf and ormqr
  CUSLVCHECK(hipsolverDnDgeqrf_bufferSize(cusolverH, m, m, d_A, lda, &lwork));
  CUDACHECK(hipMalloc((void **)&d_work, sizeof(double) * lwork));
  // step 4: compute QR factorization
  CUSLVCHECK(hipsolverDnDgeqrf(cusolverH, m, m, d_A, lda, d_tau, d_work, lwork, devInfo));
  CUDACHECK(hipDeviceSynchronize());
  // check if QR is good or not
  CUDACHECK(hipMemcpy(&info_gpu, devInfo, sizeof(int), hipMemcpyDeviceToHost));
  std::cout << "after geqrf: info_gpu = " << info_gpu << std::endl;
  assert(0 == info_gpu);
  // step 5: compute Q^T*B
  CUSLVCHECK(hipsolverDnDormqr(cusolverH, HIPBLAS_SIDE_LEFT, HIPBLAS_OP_T, m, nrhs, m, d_A, lda, d_tau,
                              d_B, ldb, d_work, lwork, devInfo));
  CUDACHECK(hipDeviceSynchronize());

  // check if QR is good or not
  CUDACHECK(hipMemcpy(&info_gpu, devInfo, sizeof(int), hipMemcpyDeviceToHost));
  std::cout << "after ormqr: info_gpu = " << info_gpu << std::endl;
  assert(0 == info_gpu);
  // step 6: compute x = R \ Q^T*B
  CUBLASCHECK(hipblasDtrsm(cublasH, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N,
                          HIPBLAS_DIAG_NON_UNIT, m, nrhs, &one, d_A, lda, d_B, ldb));
  CUDACHECK(hipDeviceSynchronize());
  CUDACHECK(hipMemcpy(XC, d_B, sizeof(double) * ldb * nrhs, hipMemcpyDeviceToHost));
  std::cout << "X = (matlab base-1)" << std::endl;
  printMatrix(m, nrhs, XC, ldb);
  // free resources
  if (d_A) hipFree(d_A);
  if (d_tau) hipFree(d_tau);
  if (d_B) hipFree(d_B);
  if (devInfo) hipFree(devInfo);
  if (d_work) hipFree(d_work);
  if (cublasH) hipblasDestroy(cublasH);
  if (cusolverH) hipsolverDnDestroy(cusolverH);
  hipDeviceReset();
  return 0;
}

