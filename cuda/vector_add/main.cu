#include <iostream>
#include <hip/hip_runtime.h>

#define CHECK(call) {                                   \
    const hipError_t error = call;                     \
    if (error != hipSuccess) {                         \
        std::cout << "Error: " << __FILE__ << ":"       \
            << __LINE__ << std::endl                    \
            << hipGetErrorString(error) << std::endl;  \
        exit(1);                                        \
    }                                                   \
}

__global__ void vectorAdd(const float *A, const float *B, float *C, int numElements) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < numElements) {
        C[i] = A[i] + B[i];
    }
}

int main(void) {
    const int numElements = 50000;
    size_t size = numElements * sizeof(float);
    std::cout << "[Vector addition of " << numElements << " elements]" << std::endl;

    float h_A[numElements];
    float h_B[numElements];
    float h_C[numElements];
    for (int i = 0; i < numElements; ++i) {
        h_A[i] = rand()/(float)RAND_MAX;
        h_B[i] = rand()/(float)RAND_MAX;
    }
    float *d_A, *d_B, *d_C;
    CHECK(hipMalloc((void **)&d_A, size));
    CHECK(hipMalloc((void **)&d_B, size));
    CHECK(hipMalloc((void **)&d_C, size));

    CHECK(hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice));

    int threadsPerBlock = 256;
    int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
    std::cout << "CUDA kernel launch with " << blocksPerGrid
            << " blocks of " << threadsPerBlock << " threads" << std::endl;
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, numElements);
    CHECK(hipGetLastError());

    std::cout << "Copy output data from the CUDA device to the host memory" << std::endl;
    CHECK(hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost));

    for (int i = 0; i < numElements; ++i) {
        if (fabs(h_A[i] + h_B[i] - h_C[i]) > 1e-5) {
            std::cerr << "Result verification failed at element " << i << std::endl;
            exit(EXIT_FAILURE);
        }
    }

    std::cout << "Test PASSED" << std::endl;

    CHECK(hipFree(d_A));
    CHECK(hipFree(d_B));
    CHECK(hipFree(d_C));

    std::cout << "Done" << std::endl;
    return 0;
}
